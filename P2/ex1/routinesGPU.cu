#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>

#include "routinesGPU.h"

#define BLOCKSIZE 16

__global__ void calculateNR(uint8_t  *im, float *NR, int height, int width) {
    // Definition
    int i, j, bi, bj;

    // Retrieve global id
    i = threadIdx.y + blockDim.y*blockIdx.y + 2;
    j = threadIdx.x + blockDim.x*blockIdx.x + 2;  

    // Retrieve id within block
    bi = threadIdx.y + 2;
    bj = threadIdx.x + 2;

    /* Shared memory preparation */
    __shared__ float im_shared[2 + int(BLOCKSIZE) + 2][2 + int(BLOCKSIZE) + 2];

    // Load left superior corner
    if (2 <= bi && bi < 4 && 2 <= bj && bj < 4){
        im_shared[bi-2][bj-2] = im[(i-2)*width + j - 2];
    }

    // Load left superior corner
    if (2 <= bi && bi < 4 && 2 <= bj && bj < 4){
        im_shared[bi-2][bj-2] = im[(i-2)*width + j - 2];
    }

    // Load right superior corner
    if (2 <= bi && bi < 4 && BLOCKSIZE <= bj && bj < BLOCKSIZE + 2){
        im_shared[bi-2][bj+2] = im[(i-2)*width + j + 2];
    }

    // Load left inferior corner
    if (BLOCKSIZE <= bi && bi <= BLOCKSIZE + 2 && 2 <= bj && bj < 4){
        im_shared[bi+2][bj-2] = im[(i+2)*width + j - 2];
    }

    // Load right inferior corner
    if (BLOCKSIZE <= bi && bi <= BLOCKSIZE + 2 && BLOCKSIZE <= bj && bj < BLOCKSIZE + 2){
        im_shared[bi+2][bj+2] = im[(i+2)*width + j + 2];
    }

    // Load superior edge
    if (2 <= bi && bi < 4){
        im_shared[bi-2][bj] = im[(i-2)*width + j];
    }

    // Load inferior edge
    if (BLOCKSIZE <= bi && bi <= BLOCKSIZE + 2){
        im_shared[bi+2][bj] = im[(i+2)*width + j];
    }

    // Load left edge
    if (2 <= bj && bj < 4){
        im_shared[bi][bj-2] = im[i*width + j - 2];
    }

    // Load right edge
    if (BLOCKSIZE <= bj && bj < BLOCKSIZE + 2){
        im_shared[bi][bj+2] = im[i*width + j + 2];
    }

    // Load center data
    im_shared[bi][bj] = im[i*width + j];

    __syncthreads();

    // Noise reduction
    if (i < height-2 && j < width - 2) 
    {
        NR[i*width+j] =
             (2.0*im_shared[bi-2][bj-2] +  4.0*im_shared[bi-2][bj-1] +  5.0*im_shared[bi-2][bj] +  4.0*im_shared[bi-2][bj+1] + 2.0*im_shared[bi-2][bj+2]
            + 4.0*im_shared[bi-1][bj-2] +  9.0*im_shared[bi-1][bj-1] + 12.0*im_shared[bi-1][bj] +  9.0*im_shared[bi-1][bj+1] + 4.0*im_shared[bi-1][bj+2]
            + 5.0*im_shared[bi  ][bj-2] + 12.0*im_shared[bi  ][bj-1] + 15.0*im_shared[bi  ][bj] + 12.0*im_shared[bi  ][bj+1] + 5.0*im_shared[bi  ][bj+2]
            + 4.0*im_shared[bi+1][bj-2] +  9.0*im_shared[bi+1][bj-1] + 12.0*im_shared[bi+1][bj] +  9.0*im_shared[bi+1][bj+1] + 4.0*im_shared[bi+1][bj+2]
            + 2.0*im_shared[bi+2][bj-2] +  4.0*im_shared[bi+2][bj-1] +  5.0*im_shared[bi+2][bj] +  4.0*im_shared[bi+2][bj+1] + 2.0*im_shared[bi+2][bj+2])
            /159.0;
    }
}


__global__ void calculateGPhi(float *NR, float *G, float *phi, float *Gx, float *Gy, int height, int width) {

	int i, j;
	float PI = 3.141593;

	j = blockIdx.x * blockDim.x + threadIdx.x;
	i = blockIdx.y * blockDim.y + threadIdx.y;
	//G[i*width+j] = 0;
	phi[i*width+j] = 0;
	if(((i >=2) && (i < height-2)) && ((j >=2) && (j < width-2))) {
		// Intensity gradient of the image
			Gx[i*width+j] = 
				 (1.0*NR[(i-2)*width+(j-2)] +  2.0*NR[(i-2)*width+(j-1)] +  (-2.0)*NR[(i-2)*width+(j+1)] + (-1.0)*NR[(i-2)*width+(j+2)]
				+ 4.0*NR[(i-1)*width+(j-2)] +  8.0*NR[(i-1)*width+(j-1)] +  (-8.0)*NR[(i-1)*width+(j+1)] + (-4.0)*NR[(i-1)*width+(j+2)]
				+ 6.0*NR[(i  )*width+(j-2)] + 12.0*NR[(i  )*width+(j-1)] + (-12.0)*NR[(i  )*width+(j+1)] + (-6.0)*NR[(i  )*width+(j+2)]
				+ 4.0*NR[(i+1)*width+(j-2)] +  8.0*NR[(i+1)*width+(j-1)] +  (-8.0)*NR[(i+1)*width+(j+1)] + (-4.0)*NR[(i+1)*width+(j+2)]
				+ 1.0*NR[(i+2)*width+(j-2)] +  2.0*NR[(i+2)*width+(j-1)] +  (-2.0)*NR[(i+2)*width+(j+1)] + (-1.0)*NR[(i+2)*width+(j+2)]);


			Gy[i*width+j] = 
				 ((-1.0)*NR[(i-2)*width+(j-2)] + (-4.0)*NR[(i-2)*width+(j-1)] +  (-6.0)*NR[(i-2)*width+(j)] + (-4.0)*NR[(i-2)*width+(j+1)] + (-1.0)*NR[(i-2)*width+(j+2)]
				+ (-2.0)*NR[(i-1)*width+(j-2)] + (-8.0)*NR[(i-1)*width+(j-1)] + (-12.0)*NR[(i-1)*width+(j)] + (-8.0)*NR[(i-1)*width+(j+1)] + (-2.0)*NR[(i-1)*width+(j+2)]
				+    2.0*NR[(i+1)*width+(j-2)] +    8.0*NR[(i+1)*width+(j-1)] +    12.0*NR[(i+1)*width+(j)] +    8.0*NR[(i+1)*width+(j+1)] +    2.0*NR[(i+1)*width+(j+2)]
				+    1.0*NR[(i+2)*width+(j-2)] +    4.0*NR[(i+2)*width+(j-1)] +     6.0*NR[(i+2)*width+(j)] +    4.0*NR[(i+2)*width+(j+1)] +    1.0*NR[(i+2)*width+(j+2)]);

			G[i*width+j]   = sqrtf((Gx[i*width+j]*Gx[i*width+j])+(Gy[i*width+j]*Gy[i*width+j]));	//G = √Gx²+Gy²
			phi[i*width+j] = atan2f(fabs(Gy[i*width+j]),fabs(Gx[i*width+j]));

			if(fabs(phi[i*width+j])<=PI/8 )
				phi[i*width+j] = 0;
			else if (fabs(phi[i*width+j])<= 3*(PI/8))
				phi[i*width+j] = 45;
			else if (fabs(phi[i*width+j]) <= 5*(PI/8))
				phi[i*width+j] = 90;
			else if (fabs(phi[i*width+j]) <= 7*(PI/8))
				phi[i*width+j] = 135;
			else phi[i*width+j] = 0;
		
	}




}

__global__ void calculatePedge(float *G, float *phi, uint8_t *pedge, int height, int width) {
	
	int i, j;

	j = blockIdx.x * blockDim.x + threadIdx.x;
	i = blockIdx.y * blockDim.y + threadIdx.y;
	pedge[i*width+j] = 0;
	if(((i >=3) && (i < height-3)) && ((j >=3) && (j < width-3))) {
		
			if(phi[i*width+j] == 0){
				if(G[i*width+j]>G[i*width+j+1] && G[i*width+j]>G[i*width+j-1]) //edge is in N-S
					pedge[i*width+j] = 1;

			} else if(phi[i*width+j] == 45) {
				if(G[i*width+j]>G[(i+1)*width+j+1] && G[i*width+j]>G[(i-1)*width+j-1]) // edge is in NW-SE
					pedge[i*width+j] = 1;

			} else if(phi[i*width+j] == 90) {
				if(G[i*width+j]>G[(i+1)*width+j] && G[i*width+j]>G[(i-1)*width+j]) //edge is in E-W
					pedge[i*width+j] = 1;

			} else if(phi[i*width+j] == 135) {
				if(G[i*width+j]>G[(i+1)*width+j-1] && G[i*width+j]>G[(i-1)*width+j+1]) // edge is in NE-SW
					pedge[i*width+j] = 1;
			}
	}
}

__global__ void calculateImageOut(uint8_t *image_out, float *G, uint8_t *pedge, float level, int height, int width) {

	float lowthres, hithres;
	int i, j;
	int ii, jj;

	j = blockIdx.x * blockDim.x + threadIdx.x;
	i = blockIdx.y * blockDim.y + threadIdx.y;

	// Hysteresis Thresholding
	lowthres = level/2;
	hithres  = 2*(level);
	image_out[i*width+j] = 0;
	if(((i >=3) && (i < height-3)) && ((j >=3) && (j < width-3))) {
		if(G[i*width+j]>hithres && pedge[i*width+j])
			image_out[i*width+j] = 255;
		else if(pedge[i*width+j] && G[i*width+j]>=lowthres && G[i*width+j]<hithres)
			// check neighbours 3x3
			for (ii=-1;ii<=1; ii++)
				for (jj=-1;jj<=1; jj++)
					if (G[(i+ii)*width+j+jj]>hithres)
						image_out[i*width+j] = 255;
	}
}

void cannyGPU(uint8_t *im, uint8_t *image_out, 
	float *NR, float *G, float *phi, float *Gx, float *Gy, uint8_t *pedge,
	float level, int height, int width)
{

	int nThreads_previo = 16;	
	dim3 nThreads(nThreads_previo, nThreads_previo);
	int myblocks;
	if (height%16==0)
		myblocks=height/16;
	else 
		myblocks = height/16+1;

	int myblocks2;
	if (width%16==0)
		myblocks2=width/16;
	else 
		myblocks2 = width/16+1;

	dim3 nBlocks(myblocks2, myblocks);

	
	calculateNR<<<nBlocks,nThreads>>>(im, NR, height, width);
	hipDeviceSynchronize();

	calculateGPhi<<<nBlocks,nThreads>>>(NR, G, phi, Gx, Gy, height, width);
	hipDeviceSynchronize();

	calculatePedge<<<nBlocks,nThreads>>>(G, phi, pedge, height, width);
	hipDeviceSynchronize();
	
	// Edge
	calculateImageOut<<<nBlocks,nThreads>>>(image_out, G, pedge, level, height, width);
}

void houghtransform(uint8_t *im, int width, int height, uint32_t *accumulators, int accu_width, int accu_height, 
	float *sin_table, float *cos_table)
{
	int i, j, theta;

	float hough_h = ((sqrt(2.0) * (float)(height>width?height:width)) / 2.0);

	for(i=0; i<accu_width*accu_height; i++)
		accumulators[i]=0;	

	float center_x = width/2.0; 
	float center_y = height/2.0;
	for(i=0;i<height;i++)  
	{  
		for(j=0;j<width;j++)  
		{  
			if( im[ (i*width) + j] > 250 ) // Pixel is edge  
			{  
				for(theta=0;theta<180;theta++)  
				{  
					float rho = ( ((float)j - center_x) * cos_table[theta]) + (((float)i - center_y) * sin_table[theta]);
					accumulators[ (int)((round(rho + hough_h) * 180.0)) + theta]++;

				} 
			} 
		} 
	}
}

void getlines(int threshold, uint32_t *accumulators, int accu_width, int accu_height, int width, int height, 
	float *sin_table, float *cos_table,
	int *x1_lines, int *y1_lines, int *x2_lines, int *y2_lines, int *lines)
{
	int rho, theta, ii, jj;
	uint32_t max;

	for(rho=0;rho<accu_height;rho++)
	{
		for(theta=0;theta<accu_width;theta++)  
		{  

			if(accumulators[(rho*accu_width) + theta] >= threshold)  
			{  
				//Is this point a local maxima (9x9)  
				max = accumulators[(rho*accu_width) + theta]; 
				for(int ii=-4;ii<=4;ii++)  
				{  
					for(int jj=-4;jj<=4;jj++)  
					{  
						if( (ii+rho>=0 && ii+rho<accu_height) && (jj+theta>=0 && jj+theta<accu_width) )  
						{  
							if( accumulators[((rho+ii)*accu_width) + (theta+jj)] > max )  
							{
								max = accumulators[((rho+ii)*accu_width) + (theta+jj)];
							}  
						}  
					}  
				}  

				if(max == accumulators[(rho*accu_width) + theta]) //local maxima
				{
					int x1, y1, x2, y2;  
					x1 = y1 = x2 = y2 = 0;  

					if(theta >= 45 && theta <= 135)  
					{
						if (theta>90) {
							//y = (r - x cos(t)) / sin(t)  
							x1 = width/2;  
							y1 = ((float)(rho-(accu_height/2)) - ((x1 - (width/2) ) * cos_table[theta])) / sin_table[theta] + (height / 2);
							x2 = width;  
							y2 = ((float)(rho-(accu_height/2)) - ((x2 - (width/2) ) * cos_table[theta])) / sin_table[theta] + (height / 2);  
						} else {
							//y = (r - x cos(t)) / sin(t)  
							x1 = 0;  
							y1 = ((float)(rho-(accu_height/2)) - ((x1 - (width/2) ) * cos_table[theta])) / sin_table[theta] + (height / 2);
							x2 = width*2/5;  
							y2 = ((float)(rho-(accu_height/2)) - ((x2 - (width/2) ) * cos_table[theta])) / sin_table[theta] + (height / 2); 
						}
					} else {
						//x = (r - y sin(t)) / cos(t);  
						y1 = 0;  
						x1 = ((float)(rho-(accu_height/2)) - ((y1 - (height/2) ) * sin_table[theta])) / cos_table[theta] + (width / 2);  
						y2 = height;  
						x2 = ((float)(rho-(accu_height/2)) - ((y2 - (height/2) ) * sin_table[theta])) / cos_table[theta] + (width / 2);  
					}
					x1_lines[*lines] = x1;
					y1_lines[*lines] = y1;
					x2_lines[*lines] = x2;
					y2_lines[*lines] = y2;
					(*lines)++;
				}
			}
		}
	}
}

void line_asist_GPU(uint8_t *im, int height, int width,
	float *sin_table, float *cos_table, 
	uint32_t *accum, int accu_height, int accu_width,
	int *x1, int *x2, int *y1, int *y2, int *nlines)
{
	int threshold;
	
	//GPU
	uint8_t *imageBW_GPU;
	hipMalloc((uint8_t **)&imageBW_GPU,sizeof(uint8_t)*width*height );
	hipMemcpy(imageBW_GPU,im,sizeof(uint8_t)*width*height,hipMemcpyHostToDevice);
	
	uint8_t *imageOUT = (uint8_t *)malloc(sizeof(uint8_t)*width*height);
	
	float *NR_GPU;
	float *G_GPU;
	float *phi_GPU;
	float *Gx_GPU;
	float *Gy_GPU;
	uint8_t *pedge_GPU;
	uint8_t*imageOUT_GPU;

	hipMalloc((float**)&NR_GPU,sizeof(float)*width*height);
	hipMalloc((float**)&G_GPU,sizeof(float)*width*height);
	hipMalloc((float**)&phi_GPU,sizeof(float)*width*height);
	hipMalloc((float**)&Gx_GPU,sizeof(float)*width*height);
	hipMalloc((float**)&Gy_GPU,sizeof(float)*width*height);
	hipMalloc((uint8_t**)&pedge_GPU,sizeof(uint8_t)*width*height);
	hipMalloc((uint8_t**)&imageOUT_GPU,sizeof(uint8_t)*width*height);

	//CANNY
	cannyGPU(imageBW_GPU, imageOUT_GPU, NR_GPU, G_GPU, phi_GPU, Gx_GPU, Gy_GPU, pedge_GPU, 1000.0, height, width);
	
	hipMemcpy(imageOUT,imageOUT_GPU,sizeof(uint8_t)*width*height,hipMemcpyDeviceToHost);
	//hough transform 
	houghtransform(imageOUT, width, height, accum, accu_width, accu_height, sin_table, cos_table);

	// WRITE IMAGE
	if (width>height) threshold = width/6;
	else threshold = height/6;


	getlines(threshold, accum, accu_width, accu_height, width, height, 
		sin_table, cos_table,
		x1, y1, x2, y2, nlines);
}
